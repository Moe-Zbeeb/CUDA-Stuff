#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

__global__ void axpb_kernel(double* x, double* y, double a, double b, unsigned int M) {

    // TODO Progress  
    int IdxThread = blockDim.x * blockIdx.x + threadIdx.x ;   
    if( IdxThread < M ){
        y[IdxThread] = a*x[IdxThread] + b; 
    }
}

void axpb_gpu(double* x, double* y, double a, double b, unsigned int M) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);

    // TODO   done 
    double *x_d, *y_d; 
    hipMalloc((void**) &x_d, M*sizeof(double));    
    hipMalloc((void**) &y_d, M*sizeof(double)); 





    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);

    // TODO  done
    hipMemcpy(x_d, x, M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, M*sizeof(double), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);

    // TODO Done 
    const unsigned int numThreadsPerBlock = 1024;    
    const unsigned int numBlocks = (M + numThreadsPerBlock - 1 )/numThreadsPerBlock; 
    axpb_kernel <<< numBlocks, numThreadsPerBlock >>> (x_d, y_d, a, b, M);




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);

    // TODO done  
    hipMemcpy(y, y_d, M*sizeof(double), hipMemcpyDeviceToHost); 


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);

    // TODO 
    hipFree(x_d);
    hipFree(y_d);  



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

